#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "ValidDim.h"
#include "vscode_cuda.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                                \
  for (int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < (n); \
       i += (blockDim.x * gridDim.x))

namespace
{

template <typename scalar_t>
__global__ void LMorphForwardCUDA(
    const size_t output_size,
    const ValidDim dim,
    const scalar_t *input,
    const scalar_t *filter,
    const scalar_t *p,
    scalar_t *lower,
    scalar_t *out)
{
  CUDA_1D_KERNEL_LOOP(index, output_size)
  {
    const size_t output_col = index % dim.output_cols;
    const size_t output_row = (index / dim.output_cols) % dim.output_rows;
    const size_t output_channel = (index / dim.output_cols / dim.output_rows) % dim.output_channels;
    const size_t batch = index / dim.output_cols / dim.output_rows / dim.output_channels;

    const size_t input_batch_offset = batch * dim.input_batch_stride;
    const size_t filter_output_offset = output_channel * dim.filter_output_stride;
    const size_t p_output_offset = output_channel * dim.input_channels;

    scalar_t upper_s = 0.0;
    scalar_t lower_s = 0.0;
    for (size_t input_channel = 0; input_channel < dim.input_channels; ++input_channel)
    {
      const size_t input_channel_offset = input_batch_offset + input_channel * dim.input_channel_stride;
      const size_t filter_channel_offset = filter_output_offset + input_channel * dim.filter_input_stride;
      scalar_t p_s = p[p_output_offset + input_channel];
      for (size_t filter_row = 0; filter_row < dim.filter_rows; ++filter_row)
      {
        const size_t input_row_offset = input_channel_offset + (output_row + filter_row) * dim.input_row_stride;
        const size_t filter_row_offset = filter_channel_offset + filter_row * dim.filter_row_stride;
        for (size_t filter_col = 0; filter_col < dim.filter_cols; ++filter_col)
        {
          scalar_t neighbor = input[input_row_offset + output_col + filter_col];
          scalar_t neighbor_filter = filter[filter_row_offset + filter_col];
          scalar_t sum = neighbor + neighbor_filter;
          scalar_t res_lower = pow(sum, p_s);
          scalar_t res_upper = res_lower * sum;
          lower_s += res_lower;
          upper_s += res_upper;
        }
      }
    }
    lower[index] = lower_s;
    out[index] = upper_s / lower_s;
  }
}

template <typename scalar_t>
__global__ void LMorphBackwardCUDA(
    const size_t output_size,
    const ValidDim dim,
    const scalar_t *grad_output,
    const scalar_t *output,
    const scalar_t *lower,
    const scalar_t *input,
    const scalar_t *filter,
    const scalar_t *p,
    scalar_t *grad_input,
    scalar_t *grad_filter,
    scalar_t *grad_p)
{
  CUDA_1D_KERNEL_LOOP(index, output_size)
  {
    const size_t output_col = index % dim.output_cols;
    const size_t output_row = (index / dim.output_cols) % dim.output_rows;
    const size_t output_channel = (index / dim.output_cols / dim.output_rows) % dim.output_channels;
    const size_t batch = index / dim.output_cols / dim.output_rows / dim.output_channels;

    scalar_t lower_s = lower[index];
    scalar_t output_s = output[index];
    scalar_t grad_output_s = grad_output[index];

    const size_t input_batch_offset = batch * dim.input_batch_stride;
    const size_t filter_output_offset = output_channel * dim.filter_output_stride;
    const size_t p_output_offset = output_channel * dim.input_channels;

    for (size_t input_channel = 0; input_channel < dim.input_channels; ++input_channel)
    {
      const size_t input_channel_offset = input_batch_offset + input_channel * dim.input_channel_stride;
      const size_t filter_input_offset = filter_output_offset + input_channel * dim.filter_input_stride;
      const size_t p_input_offset = p_output_offset + input_channel;
      scalar_t p_s = p[p_input_offset];
      scalar_t grad_p_input_s = 0.0;

      for (size_t filter_row = 0; filter_row < dim.filter_rows; ++filter_row)
      {
        const size_t input_row_offset = input_channel_offset + (output_row + filter_row) * dim.input_row_stride;
        const size_t filter_row_offset = filter_input_offset + filter_row * dim.filter_row_stride;
        for (size_t filter_col = 0; filter_col < dim.filter_cols; ++filter_col)
        {
          const size_t input_col_offset = input_row_offset + output_col + filter_col;
          const size_t filter_col_offset = filter_row_offset + filter_col;
          scalar_t sum = input[input_col_offset] + filter[filter_col_offset];
          scalar_t sum_p_m1 = pow(sum, p_s - 1.0);
          scalar_t sum_p = sum_p_m1 * sum;
          scalar_t sum_p_p1 = sum_p * sum;
          scalar_t sum_log = log(sum);

          // Be careful with the order of operations here: `upper`, `lower` and `sum` can get very big.
          // As such, they should never be multiplied together directly, lest `inf` values appear in
          // our gradient. What we do here is divide big numbers together before multiplying them again.
          scalar_t grad_input_filter_s = (p_s + 1.0) * (sum_p / lower_s) - p_s * (sum_p_m1 / lower_s) * output_s;
          grad_input_filter_s *= grad_output_s;

          scalar_t grad_p_s = sum_log * (sum_p_p1 / lower_s) - sum_log * (sum_p / lower_s) * output_s;
          grad_p_input_s += grad_p_s;

          // Atomic add: multiple kernels might be trying to concurrently operate on the same memory location,
          // which would cause operations to be dropped non-deterministically.
          atomicAdd(grad_input + input_col_offset, grad_input_filter_s);
          atomicAdd(grad_filter + filter_col_offset, grad_input_filter_s);
        }
      }

      atomicAdd(grad_p + p_input_offset, grad_p_input_s * grad_output_s);
    }
  }
}

} // namespace

std::tuple<at::Tensor, at::Tensor> LMorph_forward_cuda(
    const at::Tensor &input,
    const at::Tensor &filter,
    const at::Tensor &p)
{
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(filter.device().is_cuda(), "filter must be a CUDA tensor");
  AT_ASSERTM(p.device().is_cuda(), "p must be a CUDA tensor");

  auto dim = ValidDim(input.sizes(), filter.sizes(), p.sizes());

  at::TensorArg input_t{input, "input", 1}, filter_t{filter, "filter", 2}, p_t{p, "p", 3};

  at::CheckedFrom c = "LMorph_forward_cuda";
  at::checkAllSameGPU(c, {input_t, filter_t, p_t});
  at::checkAllSameType(c, {input_t, filter_t, p_t});

  at::cuda::CUDAGuard device_guard(input.device());

  auto output = torch::empty({input.size(0),
                              filter.size(0),
                              dim.output_rows,
                              dim.output_cols},
                             input.options());
  auto lower = torch::empty_like(output);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(output.numel()), static_cast<int64_t>(512)));
  dim3 block(512);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "LMorph_forward", ([&] {
                               LMorphForwardCUDA<scalar_t> KERNEL_ARG4(grid, block, 0, stream)(
                                   output.numel(),
                                   dim,
                                   input.data_ptr<scalar_t>(),
                                   filter.data_ptr<scalar_t>(),
                                   p.data_ptr<scalar_t>(),
                                   lower.data_ptr<scalar_t>(),
                                   output.data_ptr<scalar_t>());
                             }));
  AT_CUDA_CHECK(hipGetLastError());

  return {output, lower};
}

std::tuple<at::Tensor, at::Tensor, at::Tensor> LMorph_backward_cuda(
    const at::Tensor &grad_output,
    const at::Tensor &output,
    const at::Tensor &lower,
    const at::Tensor &input,
    const at::Tensor &filter,
    const at::Tensor &p)
{
  AT_ASSERTM(grad_output.device().is_cuda(), "grad_output must be a CUDA tensor");
  AT_ASSERTM(output.device().is_cuda(), "output must be a CUDA tensor");
  AT_ASSERTM(lower.device().is_cuda(), "lower must be a CUDA tensor");
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(filter.device().is_cuda(), "filter must be a CUDA tensor");
  AT_ASSERTM(p.device().is_cuda(), "p must be a CUDA tensor");

  auto dim = ValidDim(input.sizes(), filter.sizes(), p.sizes());

  at::TensorArg grad_output_t{grad_output, "grad_output", 1}, lower_t{lower, "lower", 2}, input_t{input, "input", 3}, filter_t{filter, "filter", 4}, p_t{p, "p", 5};

  at::CheckedFrom c = "LMorph_backward_cuda";
  at::checkAllSameGPU(c, {grad_output_t, lower_t, input_t, filter_t, p_t});
  at::checkAllSameType(c, {grad_output_t, lower_t, input_t, filter_t, p_t});

  at::cuda::CUDAGuard device_guard(grad_output.device());

  auto grad_input = at::zeros_like(input);
  auto grad_filter = at::zeros_like(filter);
  auto grad_p = at::zeros_like(p);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(grad_output.numel()), static_cast<int64_t>(512)));
  dim3 block(512);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "LMorph_backward", ([&] {
                               LMorphBackwardCUDA<scalar_t> KERNEL_ARG4(grid, block, 0, stream)(
                                   grad_output.numel(),
                                   dim,
                                   grad_output.data_ptr<scalar_t>(),
                                   output.data_ptr<scalar_t>(),
                                   lower.data_ptr<scalar_t>(),
                                   input.data_ptr<scalar_t>(),
                                   filter.data_ptr<scalar_t>(),
                                   p.data_ptr<scalar_t>(),
                                   grad_input.data_ptr<scalar_t>(),
                                   grad_filter.data_ptr<scalar_t>(),
                                   grad_p.data_ptr<scalar_t>());
                             }));
  AT_CUDA_CHECK(hipGetLastError());

  return {grad_input, grad_filter, grad_p};
}
